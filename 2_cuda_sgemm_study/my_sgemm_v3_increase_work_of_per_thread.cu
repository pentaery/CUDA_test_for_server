
#include <hip/hip_runtime.h>
#include <cstdio>

#define A(i, j) a[i * n + j]
#define B(i, j) b[i * n + j]
void random_matrix(int m, int n, float *a) {
  int i, j;
  for (i = 0; i < m; ++i) {
    for (j = 0; j < n; ++j) {
#if 1
      A(i, j) = 2.0 * (float)drand48() - 1.0;
#else
      A(i, j) = (j - i) % 3;
#endif
    }
  }
}

float compare_matrices(int m, int n, float *a, float *b) {
  int i, j;
  float max_diff = 0.0, diff;
  int printed = 0;

  for (i = 0; i < m; ++i) {
    for (j = 0; j < n; ++j) {
      max_diff = abs(A(i, j) - B(i, j));
      if (printed == 0) {
        if (max_diff > 0.5f || max_diff < -0.5f) {
          printf("\n error: i %d j %d diff %f  got %f expect %f \n", i, j,
                 max_diff, A(i, j), B(i, j));
          printed = 1;
        }
      }
    }
  }
  return max_diff;
}

void cpu_sgemm(float *A_ptr, float *B_ptr, float *C_ptr, const int M,
               const int N, const int K) {
  for (int m = 0; m < M; ++m) {
    for (int n = 0; n < N; ++n) {
      float sum = 0.f;
      for (int k = 0; k < K; ++k) {
        sum += A_ptr[m * K + k] * B_ptr[k * N + n];
      }
      C_ptr[m * N + n] = sum;
    }
  }
}

template <unsigned int BLOCK_SIZE, unsigned int STRIDE>
__global__ void cuda_sgemm(float *A_ptr, float *B_ptr, float *C_ptr,
                           const int M, const int N, const int K) {

  float *A_ptr_start = A_ptr + STRIDE * blockIdx.x * blockDim.x * K;
  float *B_ptr_start = B_ptr + STRIDE * blockIdx.y * blockDim.y;

  __shared__ float a_shared[BLOCK_SIZE * STRIDE][BLOCK_SIZE * STRIDE];
  __shared__ float b_shared[BLOCK_SIZE * STRIDE][BLOCK_SIZE * STRIDE];

  float temp[STRIDE][STRIDE] = {0.f};

  for (int s = 0; s < K; s += blockDim.x * STRIDE) {

    for (int i = 0; i < STRIDE; i++) {
      for (int j = 0; j < STRIDE; j++) {
        a_shared[threadIdx.x + i * BLOCK_SIZE][threadIdx.y + j * BLOCK_SIZE] =
            A_ptr_start[(threadIdx.x + i * BLOCK_SIZE) * K + threadIdx.y + s +
                        j * BLOCK_SIZE];
        b_shared[threadIdx.x + i * BLOCK_SIZE][threadIdx.y + j * BLOCK_SIZE] =
            B_ptr_start[(threadIdx.x + i * BLOCK_SIZE + s) * N + threadIdx.y +
                        j * BLOCK_SIZE];
      }
    }
    __syncthreads();

    for (int i = 0; i < STRIDE; i++) {
      for (int j = 0; j < STRIDE; j++) {
        for (int k = 0; k < BLOCK_SIZE * STRIDE; ++k) {
          temp[i][j] +=
              a_shared[threadIdx.x + i * BLOCK_SIZE][k] *
              b_shared[k][threadIdx.y + j * BLOCK_SIZE];
        }
      }
    }
    __syncthreads();
  }

  float *C_ptr_start = C_ptr + (blockIdx.x * blockDim.x * STRIDE) * N +
                       blockIdx.y * blockDim.y * STRIDE;
  for (int i = 0; i < STRIDE; i++) {
    for (int j = 0; j < STRIDE; j++) {
      C_ptr_start[(threadIdx.x + i * BLOCK_SIZE) * N + threadIdx.y +
                  j * BLOCK_SIZE] = temp[i][j];
    }
  }
}

int main() {
  int m = 1024;
  int n = 1024;
  int k = 1024;

  const size_t mem_size_A = m * k * sizeof(float);
  const size_t mem_size_B = k * n * sizeof(float);
  const size_t mem_size_C = m * n * sizeof(float);

  float *matrix_A_host = (float *)malloc(mem_size_A);
  float *matrix_B_host = (float *)malloc(mem_size_B);

  float *matrix_C_host_gpu_calc = (float *)malloc(mem_size_C);
  float *matrix_C_host_cpu_calc = (float *)malloc(mem_size_C);

  random_matrix(m, k, matrix_A_host);
  random_matrix(k, n, matrix_B_host);
  memset(matrix_C_host_gpu_calc, 0, mem_size_C);
  memset(matrix_C_host_cpu_calc, 0, mem_size_C);

  float *matrix_A_device, *matrix_B_device, *matrix_C_device;

  hipMalloc((void **)&matrix_A_device, mem_size_A);
  hipMalloc((void **)&matrix_B_device, mem_size_B);
  hipMalloc((void **)&matrix_C_device, mem_size_C);

  hipMemcpy(matrix_A_device, matrix_A_host, mem_size_A,
             hipMemcpyHostToDevice);
  hipMemcpy(matrix_B_device, matrix_B_host, mem_size_B,
             hipMemcpyHostToDevice);

  cpu_sgemm(matrix_A_host, matrix_B_host, matrix_C_host_cpu_calc, m, n, k);

  constexpr int BLOCK = 16;
  constexpr int STRIDE = 4;
  dim3 block(BLOCK, BLOCK);
  dim3 grid((m + BLOCK - 1) / BLOCK / STRIDE, (n + BLOCK - 1) / BLOCK / STRIDE);
  cuda_sgemm<BLOCK, STRIDE><<<grid, block>>>(matrix_A_device, matrix_B_device,
                                             matrix_C_device, m, n, k);

  hipMemcpy(matrix_C_host_gpu_calc, matrix_C_device, mem_size_C,
             hipMemcpyDeviceToHost);

  float diff =
      compare_matrices(m, n, matrix_C_host_cpu_calc, matrix_C_host_gpu_calc);

  if (diff > 0.5f || diff < -0.5f) {
    printf("Error.\n");
    exit(-1);
  } else {
    printf("Success\n");
  }

  free(matrix_A_host);
  free(matrix_B_host);
  free(matrix_C_host_gpu_calc);
  free(matrix_C_host_cpu_calc);

  hipFree(matrix_A_device);
  hipFree(matrix_B_device);
  hipFree(matrix_C_device);

  return 0;
}