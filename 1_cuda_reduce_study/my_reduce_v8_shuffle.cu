#include <cstdio>
#include <hip/hip_runtime.h>

#include <stdlib.h>

#define THREAD_PER_BLOCK 256

template <unsigned int NUM_PER_BLOCK, unsigned int NUM_PER_THREAD>
__global__ void reduce(float *d_input, float *d_output) {
  float sum = 0.f;
  float *begin = &d_input[blockIdx.x * NUM_PER_BLOCK];

  for (int i = 0; i < NUM_PER_THREAD; ++i) {
    sum += begin[threadIdx.x + i * THREAD_PER_BLOCK];
  }

  sum += __shfl_down_sync(0xffffffff, sum, 16);
  sum += __shfl_down_sync(0xffffffff, sum, 8);
  sum += __shfl_down_sync(0xffffffff, sum, 4);
  sum += __shfl_down_sync(0xffffffff, sum, 2);
  sum += __shfl_down_sync(0xffffffff, sum, 1);

  __shared__ float warpLevelSums[32];

  const int laneId = threadIdx.x % 32;
  const int warpId = threadIdx.x / 32;

  if (laneId == 0) {
    warpLevelSums[warpId] = sum;
  }
  __syncthreads();

  if (warpId == 0) {
    sum = (laneId < blockDim.x / 32) ? warpLevelSums[laneId] : 0.f;
    sum += __shfl_down_sync(0xffffffff, sum, 16);
    sum += __shfl_down_sync(0xffffffff, sum, 8);
    sum += __shfl_down_sync(0xffffffff, sum, 4);
    sum += __shfl_down_sync(0xffffffff, sum, 2);
    sum += __shfl_down_sync(0xffffffff, sum, 1);
  }

  if (threadIdx.x == 0) {
    d_output[blockIdx.x] = sum;
  } 

}

bool check(float *output, float *result, int block_num) {
  for (int i = 0; i < block_num; ++i) {
    if (abs(output[i] - result[i]) > 5e-3) {
      return false;
    }
  }
  return true;
}

int main() {

  const int N = 32 * 1024 * 1024;
  float *input = (float *)malloc(N * sizeof(float));
  float *d_input;
  hipMalloc((void **)&d_input, N * sizeof(float));

  constexpr int block_num = 1024;
  constexpr int num_per_block = N / block_num;
  constexpr int num_per_thread = num_per_block / THREAD_PER_BLOCK;
  float *output = (float *)malloc(block_num * sizeof(float));
  float *d_output;
  hipMalloc((void **)&d_output, block_num * sizeof(float));
  float *result = (float *)malloc(block_num * sizeof(float));

  for (int i = 0; i < N; ++i) {
    input[i] = 2.0 * (float)drand48() - 1.0;
  }

  for (int i = 0; i < block_num; ++i) {
    float cur = 0;
    for (int j = 0; j < num_per_block; ++j) {
      cur += input[i * num_per_block + j];
    }
    result[i] = cur;
  }

  hipMemcpy(d_input, input, N * sizeof(float), hipMemcpyHostToDevice);

  dim3 Grid(block_num, 1);
  dim3 Block(THREAD_PER_BLOCK, 1);

  reduce<num_per_block, num_per_thread><<<Grid, Block>>>(d_input, d_output);

  hipMemcpy(output, d_output, block_num * sizeof(float),
             hipMemcpyDeviceToHost);

  if (check(output, result, block_num)) {
    printf("Success!\n");
  } else {
    printf("Failed\n");
    for (int i = 0; i < block_num; ++i) {
      printf("%lf\n", output[i] - result[i]);
    }
  }

  hipFree(d_input);
  hipFree(d_output);
  return 0;
  
}