#include <cstdio>
#include <hip/hip_runtime.h>

#include <stdlib.h>
#include <chrono>

#define THREAD_PER_BLOCK 256

__global__ void reduce(float *d_input, float *d_output) {
  __shared__ float shared[THREAD_PER_BLOCK];
  float *begin = &d_input[blockIdx.x * blockDim.x];
  shared[threadIdx.x] = begin[threadIdx.x];
  __syncthreads();
  for (int i = 1; i < blockDim.x; i *= 2) {
    if (threadIdx.x * (2 * i) < blockDim.x) {
      int index = threadIdx.x * (2 * i);
      shared[index] += shared[index + i];
    }
    __syncthreads();
  }
  if (threadIdx.x == 0) {
    d_output[blockIdx.x] = shared[0];
  }
}

bool check(float *output, float *result, int block_num) {
  for (int i = 0; i < block_num; ++i) {
    if (abs(output[i] - result[i]) > 1e-4) {
      return false;
    }
  }
  return true;
}

int main() {

  const int N = 32 * 1024 * 1024;
  float *input = (float *)malloc(N * sizeof(float));
  float *d_input;
  hipMalloc((void **)&d_input, N * sizeof(float));

  int block_num = N / THREAD_PER_BLOCK;
  float *output = (float *)malloc(block_num * sizeof(float));
  float *d_output;
  hipMalloc((void **)&d_output, block_num * sizeof(float));
  float *result = (float *)malloc(block_num * sizeof(float));

  for (int i = 0; i < N; ++i) {
    input[i] = 2.0 * (float)drand48() - 1.0;
  }

  for (int i = 0; i < block_num; ++i) {
    float cur = 0;
    for (int j = 0; j < THREAD_PER_BLOCK; ++j) {
      cur += input[i * THREAD_PER_BLOCK + j];
    }
    result[i] = cur;
  }

  hipMemcpy(d_input, input, N * sizeof(float), hipMemcpyHostToDevice);

  dim3 Grid(N / THREAD_PER_BLOCK, 1);
  dim3 Block(THREAD_PER_BLOCK, 1);
  auto begin = std::chrono::high_resolution_clock::now();
  reduce<<<Grid, Block>>>(d_input, d_output);
  auto end = std::chrono::high_resolution_clock::now();
  auto elapsed =
      std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin);
  printf("Time measured: %.7f seconds.\n", elapsed.count() * 1e-9);
  hipMemcpy(output, d_output, block_num * sizeof(float),
             hipMemcpyDeviceToHost);

  if (check(output, result, block_num)) {
    printf("Success\n");
  } else {
    printf("Failed\n");
    for (int i = 0; i < block_num; ++i) {
      printf("%lf\n", output[i] - result[i]);
    }
  }

  hipFree(d_input);
  hipFree(d_output);
  return 0;
}