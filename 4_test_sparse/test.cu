#include <cassert>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <iostream>

#define CHECK_CUDA(func)                                                       \
  {                                                                            \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
      std::cerr << "CUDA Error: " << hipGetErrorString(status) << " at "      \
                << __FILE__ << ":" << __LINE__ << std::endl;                   \
      assert(false);                                                           \
    }                                                                          \
  }

#define CHECK_CUSPARSE(func)                                                   \
  {                                                                            \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
      std::cerr << "cuSPARSE Error: " << status << " at " << __FILE__ << ":"   \
                << __LINE__ << std::endl;                                      \
      assert(false);                                                           \
    }                                                                          \
  }

int main() {
  // 初始化 cuSPARSE
  hipsparseHandle_t handle;
  CHECK_CUSPARSE(hipsparseCreate(&handle));

  // 定义稀疏矩阵 (4x4) 和向量
  int m = 4, n = 4, nnz = 6;
  float h_csrValA[] = {1, 2, 3, 4, 5, 6};  // 非零元素值
  int h_csrRowPtrA[] = {0, 2, 3, 5, 6};    // 行指针
  int h_csrColIndA[] = {0, 3, 2, 0, 3, 1}; // 列索引
  float h_x[] = {1, 1, 1, 1};              // 输入向量
  float h_y[m] = {0};                      // 输出向量

  // 分配设备内存
  float *d_csrValA, *d_x, *d_y;
  int *d_csrRowPtrA, *d_csrColIndA;
  CHECK_CUDA(hipMalloc((void **)&d_csrValA, nnz * sizeof(float)));
  CHECK_CUDA(hipMalloc((void **)&d_csrRowPtrA, (m + 1) * sizeof(int)));
  CHECK_CUDA(hipMalloc((void **)&d_csrColIndA, nnz * sizeof(int)));
  CHECK_CUDA(hipMalloc((void **)&d_x, n * sizeof(float)));
  CHECK_CUDA(hipMalloc((void **)&d_y, m * sizeof(float)));

  // 拷贝数据到设备
  CHECK_CUDA(hipMemcpy(d_csrValA, h_csrValA, nnz * sizeof(float),
                        hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_csrRowPtrA, h_csrRowPtrA, (m + 1) * sizeof(int),
                        hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_csrColIndA, h_csrColIndA, nnz * sizeof(int),
                        hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_x, h_x, n * sizeof(float), hipMemcpyHostToDevice));

  // 创建稀疏矩阵描述符
  hipsparseSpMatDescr_t matA;
  CHECK_CUSPARSE(hipsparseCreateCsr(&matA, m, n, nnz, d_csrRowPtrA, d_csrColIndA,
                                   d_csrValA, HIPSPARSE_INDEX_32I,
                                   HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO,
                                   HIP_R_32F));

  // 创建向量描述符
  hipsparseDnVecDescr_t vecX, vecY;
  CHECK_CUSPARSE(hipsparseCreateDnVec(&vecX, n, d_x, HIP_R_32F));
  CHECK_CUSPARSE(hipsparseCreateDnVec(&vecY, m, d_y, HIP_R_32F));

  // 计算所需的外部缓冲区大小
  size_t bufferSize = 0;
  float alpha = 1.0f, beta = 0.0f;
  CHECK_CUSPARSE(hipsparseSpMV_bufferSize(
      handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecX, &beta, vecY,
      HIP_R_32F, HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize));

  // 分配外部缓冲区
  void *dBuffer = nullptr;
  CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));

  // 执行稀疏矩阵-向量乘法 (SpMV)
  CHECK_CUSPARSE(hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha,
                              matA, vecX, &beta, vecY, HIP_R_32F,
                              HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer));

  // 拷贝结果回主机
  CHECK_CUDA(hipMemcpy(h_y, d_y, m * sizeof(float), hipMemcpyDeviceToHost));

  // 打印结果
  std::cout << "Result y: ";
  for (int i = 0; i < m; i++) {
    std::cout << h_y[i] << " ";
  }
  std::cout << std::endl;

  // 释放资源
  CHECK_CUDA(hipFree(d_csrValA));
  CHECK_CUDA(hipFree(d_csrRowPtrA));
  CHECK_CUDA(hipFree(d_csrColIndA));
  CHECK_CUDA(hipFree(d_x));
  CHECK_CUDA(hipFree(d_y));
  CHECK_CUDA(hipFree(dBuffer));
  CHECK_CUSPARSE(hipsparseDestroySpMat(matA));
  CHECK_CUSPARSE(hipsparseDestroyDnVec(vecX));
  CHECK_CUSPARSE(hipsparseDestroyDnVec(vecY));
  CHECK_CUSPARSE(hipsparseDestroy(handle));

  return 0;
}